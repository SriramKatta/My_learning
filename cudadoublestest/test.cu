
#include <hip/hip_runtime.h>
#include <stdio.h>

#define prinlnloc printf("Line %d\n", __LINE__)

int main(int argc, char const *argv[])
{
    double **dev;
    hipHostMalloc(&dev, 10 * sizeof(double*), hipHostMallocDefault);
    prinlnloc;
    for (int i = 0; i < 10; i++) {
        hipMalloc((void**)&dev[i], 10 * sizeof(double));
        prinlnloc;
    }
    
    prinlnloc;


    for (int i = 0; i < 10; i++) {
        hipFree(dev[i]);
    }
    hipHostFree(dev);

    return 0;
}
